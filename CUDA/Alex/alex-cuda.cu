
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 25;
const int blocksize = 25;

__global__
void hello(char *a)
{
 
}

int main()
{
 char a[N] = "Hello World, for Dr. Li!";
 

 char *ad;
 
 const int csize = N*sizeof(char);


 printf("%s\n", a);

 hipMalloc( (void**)&ad, csize );

 hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
 dim3 dimBlock(blocksize, 1 );
 dim3 dimGrid( 1, 1 );
 hello<<<dimGrid, dimBlock>>>(ad);
 hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
 hipFree( ad );

 printf("%s\n", a);
 return EXIT_SUCCESS;
}
